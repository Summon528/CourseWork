/**********************************************************************
* DESCRIPTION:
*   Serial Concurrent Wave Equation - C Version
*   This program implements the concurrent wave equation
*********************************************************************/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define FAC 6.28318530f

void check_param(void);
void printfinal(float *);

int nsteps,                  /* number of time steps */
    tpoints;                 /* total points along string */

/**********************************************************************
*	Checks input values from parameters
*********************************************************************/
void check_param(void) {
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: ",
            MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", MINPOINTS,
                MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

void init_line(float *values) {
    int j;
    for (j = 1; j <= tpoints; j++) {
        values[j] = sin(FAC * ((float)(j - 1) / (tpoints - 1)));
    }
}

__global__
void work(float *values, int nsteps, int tpoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (idx > tpoints) return;
    float oldvalt, newvalt, valt;
    oldvalt = valt = values[idx];
    for (int i = 1; i <= nsteps; i++) {
        /* global endpoints */
        if (idx == 1 || idx == tpoints) {
            newvalt = 0.0f;
        } else {
            newvalt = (2.0 * valt) - oldvalt + ((double)-0.18f * valt);
        }
        oldvalt = valt;
        valt = newvalt;
    }
    values[idx] = valt;
}

/**********************************************************************
*     Print final results
*********************************************************************/
void printfinal(float *values) {
    int i;
    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i % 10 == 0) printf("\n");
    }
}

/**********************************************************************
*	Main program
*********************************************************************/
int main(int argc, char *argv[]) {
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param();
    float *values;
    hipMallocManaged (&values, sizeof(float) * (tpoints + 1));
    printf("Initializing points on the line...\n");
    init_line(values);
    printf("Updating all points for all time steps...\n");
    int blockSize = 1024;
    int numBlocks = (tpoints + blockSize - 1) / blockSize;
    work<<<numBlocks, blockSize>>>(values, nsteps, tpoints);
    hipDeviceSynchronize();
    printf("Printing final results...\n");
    printfinal(values);
    printf("\nDone.\n\n");

    return 0;
}
